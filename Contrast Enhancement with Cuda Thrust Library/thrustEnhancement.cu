#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <chrono>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include "nppEnhancement.cuh"

struct transformFunction 
{
	Npp8u min, max;

	transformFunction(Npp8u _min, Npp8u _max) {
		min = _min;
		max = _max;
	}

	// The function applied on GPU.
	// Reference operator is important. Otherwise we need another memory space to copy the result. Now we apply changes on input.
	__device__ void operator()(Npp8u &pixelValue) {
		pixelValue = (pixelValue - min) * 255 / (max - min);
	}
};

int main()
{
	// Runs contrast enhancement with Thrush or NPP implementation.
	// Both implementation measures its running time.
	bool runWithThrust = true;

	if (runWithThrust) {

		Npp8u* img_Host;
		int  nWidth, nHeight, nMaxGray;

		// Load image to the host.
		std::cout << "Load PGM file." << std::endl;
		img_Host = LoadPGM("lena_before.pgm", nWidth, nHeight, nMaxGray);

		// Performance clock start (just before using GPU)
		auto start = std::chrono::steady_clock::now();

		// Load image onto GPU
		thrust::device_vector<Npp8u> img_Device(img_Host, img_Host + (nWidth * nHeight));

		// Finds minimum and maximum in vector
		// Note: It is more efficient than using min_element and max_element seperately.
		thrust::pair<thrust::device_vector<Npp8u>::iterator, thrust::device_vector<Npp8u>::iterator> minmax = thrust::minmax_element(img_Device.begin(), img_Device.end());

		// Transform each element of image by the operator() in transformFunction.
		// It basically applys contrast enhancement formula.
		thrust::for_each(img_Device.begin(), img_Device.end(), transformFunction(*minmax.first, *minmax.second));

		// Copy transformed image into host vector.
		// Then convert vector into raw host pointer.
		thrust::host_vector<Npp8u> out_Host(img_Device);
		Npp8u* out_ptr_Host = out_Host.data();

		// Performance clock stop (just after getting data from GPU)
		auto end = std::chrono::steady_clock::now();
		auto diff = end - start;
		auto diff_sec = std::chrono::duration_cast<std::chrono::milliseconds>(diff);
		std::cout << "Computation time(ms): " << diff_sec.count() << std::endl;

		// Write result.
		WritePGM("lena_after_thrust.pgm", out_ptr_Host, nWidth, nHeight, nMaxGray);
	}
	else {
		nppEnhancement();
	}

    return 0;
}

