#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include "nppEnhancement.cuh"

struct transformFunction : thrust::unary_function<Npp8u, Npp8u>
{
	Npp8u min, max;

	transformFunction(Npp8u _min, Npp8u _max) {
		min = _min;
		max = _max;
	}

	__host__ __device__
	void operator()(Npp8u pixelValue) {
		pixelValue = (pixelValue - min) * 255 / (max - min);
	}
};

int main()
{
	Npp8u* img_Host;
	int  nWidth, nHeight, nMaxGray;

	// Load image to the host.
	std::cout << "Load PGM file." << std::endl;
	img_Host = LoadPGM("lena_before.pgm", nWidth, nHeight, nMaxGray);
	
	thrust::device_vector<Npp8u> img_Device(img_Host, img_Host + (nWidth * nHeight));
	
	// Finds minimum and maximum in vector
	// Note: It is more efficient than using min_element and max_element seperately.
	thrust::pair<thrust::device_vector<Npp8u>::iterator, thrust::device_vector<Npp8u>::iterator> minmax = thrust::minmax_element(img_Device.begin(), img_Device.end());

	std::cout << int(*(minmax.first)) << " - " << int(*(minmax.second)) << std::endl;

	transformFunction func(*minmax.first, *minmax.second);

	thrust::for_each(img_Device.begin(), img_Device.end(), func);

	//thrust::host_vector<Npp8u> out_Host(img_Device);

    return 0;
}

